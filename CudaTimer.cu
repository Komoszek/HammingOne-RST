#include "CudaTimer.cuh"

struct PrivateTimingCUDA{
    hipEvent_t start; 
    hipEvent_t stop;
};

CudaTimer::CudaTimer(){
    milliseconds = 0.0f;
    timings = new PrivateTimingCUDA;
    hipEventCreate(&timings->start);
    hipEventCreate(&timings->stop);
}

CudaTimer::~CudaTimer(){
    hipEventDestroy(timings->start);
    hipEventDestroy(timings->stop);
    delete timings;
}

void CudaTimer::Start(){
    milliseconds = 0.0f;
    hipEventRecord(timings->start);
}

void CudaTimer::Stop(){
    hipEventRecord(timings->stop);
    hipEventSynchronize(timings->stop);
    hipEventElapsedTime(&milliseconds, timings->start, timings->stop);
}

float CudaTimer::GetElapsedTime() {
    return milliseconds;
}